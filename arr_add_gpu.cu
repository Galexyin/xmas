
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <sys/time.h>

float timedifference_msec(struct timeval t0, struct timeval t1)
{
    return (t1.tv_sec - t0.tv_sec) * 1000.0f + (t1.tv_usec - t0.tv_usec) / 1000.0f;
}

__global__ void addArray(long long n,float* ga,float *gb){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < n)
        ga[id] += gb[id];
}
int main(void)
{
    struct timeval t0;
    struct timeval t1;
    struct timeval t2;
    struct timeval t3;
    float elapsed,processing_time;
    
    gettimeofday(&t0, 0);

    unsigned long long int n = 1000000000;
    float *a,*b,*ga,*gb;
    a = (float*)malloc(n*sizeof(float));
    b = (float*)malloc(n*sizeof(float));

    for (unsigned long long int i = 0; i < n; i++)
    {
        b[i] = 1.1f;
        a[i] = 2.2f;
    }

    
    hipMalloc(&ga,n*sizeof(float));
    hipMalloc(&gb,n*sizeof(float));
    
    hipMemcpy(ga,a,n*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(gb,b,n*sizeof(float),hipMemcpyHostToDevice);
    int blockSize = 2;
    int gridSize = (int)ceil((float)n/blockSize);
    
    gettimeofday(&t1, 0);
    addArray<<<gridSize,blockSize>>>(n,ga,gb);
    gettimeofday(&t2, 0);
    
    hipMemcpy(a,ga,n*sizeof(float),hipMemcpyDeviceToHost);
    
    free(a);
    free(b);
    hipFree(ga);
    hipFree(gb);

    gettimeofday(&t3, 0);
    elapsed = timedifference_msec(t0, t3);
    processing_time = timedifference_msec(t1, t2);

    printf("\nTotal Time Elasped:%f \nProcessing Time:%f \n\n",elapsed,processing_time);
    return 0;
}