
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <sys/time.h>

float timedifference_msec(struct timeval t0, struct timeval t1)
{
    return (t1.tv_sec - t0.tv_sec) * 1000.0f + (t1.tv_usec - t0.tv_usec) / 1000.0f;
}

int main(int argc, char const *argv[])
{

    struct timeval t0;
    struct timeval t1;
    struct timeval t2;
    struct timeval t3;
    float elapsed,processing_time;

    gettimeofday(&t0, 0);

    unsigned long long int n = 1000000000;
    float *a,*b;
    a = (float*)malloc(n*sizeof(float));
    b = (float*)malloc(n*sizeof(float));

    for (unsigned long long int i = 0; i < n; i++)
    {
        b[i] = 1.1f;
        a[i] = 2.2f;
    }

    
    gettimeofday(&t1, 0);
    for (unsigned long long int i = 0; i < n; i++)
    {
        b[i] += a[i];
    }
    gettimeofday(&t2, 0);
    free(a);
    free(b);

    gettimeofday(&t3, 0);

    elapsed = timedifference_msec(t0, t3);
    processing_time = timedifference_msec(t1, t2);

    printf("\nTotal Time Elasped:%f \nProcessing Time:%f\n\n",elapsed,processing_time);
    return 0;
}
